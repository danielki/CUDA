#include "device.h"
#include <fstream>
#include <time.h>
#include <stdio.h>

void   copyToDevice(const primitives &p, float &elapsedTime)
       {
       hipEvent_t start, stop;
	   hipEventCreate(&start);
	   hipEventCreate(&stop);
	   size_t sizeInBytes = sizeof(p);
	   primitives* devicePointer;
	   hipMalloc(&devicePointer, sizeInBytes);
	   hipEventRecord(start, 0);
	   hipMemcpy(devicePointer, &p, sizeInBytes, hipMemcpyHostToDevice);
	   hipEventRecord(stop, 0);
	   hipEventElapsedTime(&elapsedTime, start, stop);
	   hipEventDestroy(start);
	   hipEventDestroy(stop);
	   hipFree(devicePointer);
	   }
	
